#include "hip/hip_runtime.h"
﻿//Сложная 14.	Найти дисперсию
#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include <hip/device_functions.h>

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

const int maxThreadsPerBlock = 256;
const int blocksPerGrid = 32;

__global__ void sumOfSquares(float* numbers, float* result, int n, float x)
{
	__shared__ float cache[maxThreadsPerBlock];
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int cacheId = threadIdx.x;
	int totalNumberOfThreads = gridDim.x * blockDim.x;

	float tmp = 0;
	while (idx < n)
	{
		tmp += (numbers[idx] - x) * (numbers[idx] - x);
		idx += totalNumberOfThreads;
	}

	cache[cacheId] = tmp;
	__syncthreads();

	int i = blockDim.x / 2;
	while (i != 0)
	{
		if (cacheId < i)
			cache[cacheId] += cache[cacheId + i];
		__syncthreads();
		i /= 2;
	}
	if (cacheId == 0)
		result[blockIdx.x] = cache[0];
}

void main()
{
	int n;
	printf("Print count numbers: \n");
	while ((scanf("%d", &n)) != 1) {
		printf("Incorrect! Try again: ");
		while (getchar() != '\n')
			;
	}

	float x = 0;
	float* a, * b, c, * partial_c;
	float* dev_a, * dev_b, * dev_partial_c;
	a = (float*)malloc(n * sizeof(float));
	b = (float*)malloc(n * sizeof(float));
	partial_c = (float*)malloc(blocksPerGrid * sizeof(float));

	if (hipMalloc(&dev_a, n * sizeof(float)) != hipSuccess)
		printf("Error: hipMalloc");
	if (hipMalloc(&dev_b, n * sizeof(float)) != hipSuccess)
		printf("Error: hipMalloc");
	if (hipMalloc(&dev_partial_c, blocksPerGrid * sizeof(float)) != hipSuccess)
		printf("Error: hipMalloc");

	printf("Print %d numbers: \n", n);
	for (int i = 0; i < n; i++)
	{
		float tmp = 0;
		while ((scanf("%f", &tmp)) != 1) {
			printf("Incorrect! Try again: ");
			while (getchar() != '\n')
				;
		}
		x += tmp;
		a[i] = tmp;
	}

	//for (int i = 0; i < n; i++) {
	//	a[i] = i;
	//	b[i] = i * 2;
	//}

	if (hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice) != hipSuccess)
		printf("Error: hipMemcpy!");
	if (hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice) != hipSuccess)
		printf("Error: hipMemcpy!");

	sumOfSquares << <blocksPerGrid, maxThreadsPerBlock >> > (dev_a, dev_partial_c, n, (float)x / n);
	if (hipGetLastError() != hipSuccess)
		printf("Error: sumOfSquares");

	if (hipMemcpy(partial_c, dev_partial_c, sizeof(float) * blocksPerGrid, hipMemcpyDeviceToHost) != hipSuccess)
		printf("Error: hipMemcpy");

	c = 0;
	for (int i = 0; i < blocksPerGrid; i++)
	{
		c += partial_c[i];
	}
	printf("Answer: %f", c / n);

}
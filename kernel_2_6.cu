#include "hip/hip_runtime.h"
﻿#ifndef __HIPCC__ 
#define __HIPCC__
#endif

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <hip/device_functions.h>


#define N 30
#define SIZE 256
#define MIN 0
#define MAX 100
#define DIF (MAX - MIN)

__global__ void kernelFirst(hiprandState* state, unsigned long seed, int n)
{
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    if (id < n)
        hiprand_init(seed, id, 0, &state[id]);
}

__global__ void kernelGenerate(hiprandState* gState, float* result, int count)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < count) {
        hiprandState localState = gState[idx];
        float RANDOM = hiprand_uniform(&localState);
        gState[idx] = localState;
        result[idx] = RANDOM;
    }
}

void cpuGenetate(float* result, int count)
{
    for (int i = 0; i < count; i++) 
    {
        result[i] = MIN + rand() % MAX + 1;
    }
}

int main(int argc, char** argv)
{
    srand(time(NULL));
    float* dev_result;
    float* result = (float*)malloc(DIF * sizeof(float));
    float* resultCPU = (float*)malloc(DIF * sizeof(float));
    hipEvent_t start, stop;
    float gpuTime = 0.0f;
    hiprandState* devStates;


    if (hipMalloc(&devStates, DIF * sizeof(hiprandState)))
        printf("Error: cudaMalloc1\n");
    if (hipMalloc(&dev_result, DIF * sizeof(float)))
        printf("Error: cudaMalloc2\n");

    kernelFirst << < (DIF + SIZE - 1) / SIZE, SIZE >> > (devStates, time(NULL), DIF);
    if (hipGetLastError() != hipSuccess) printf("Error: Kernel");

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    kernelGenerate << < (DIF + SIZE - 1) / SIZE, SIZE >> > (devStates, dev_result, DIF);
    if (hipGetLastError() != hipSuccess) printf("Error: Kernel");

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuTime, start, stop);
    printf("Time GPU = %.2f millseconds\n", gpuTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    int start2, time2;
    start2 = clock();

    cpuGenetate(resultCPU, DIF);

    time2 = clock() - start2;
    printf("Time CPU = %d millseconds\n", time2);

    if (hipMemcpy(result, dev_result, sizeof(float) *DIF, hipMemcpyDeviceToHost) != hipSuccess)
        printf("Error: cudaMemcpy3\n");
    printf("Random numbers: \n");
    for (int i = 0; i < N; i++)
        printf("@-> %f\n", result[i]*DIF + MIN);



    return 0;
}